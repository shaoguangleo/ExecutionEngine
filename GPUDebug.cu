
#include <hip/hip_runtime.h>
template <int T>
__global__ void debugMark() {
};

#ifdef __DEBUG
void debugMark(int t) {
   switch(t) {
      case 0: debugMark<0><<<1,1>>>(); break;
      case 1: debugMark<1><<<1,1>>>(); break;
      case 2: debugMark<2><<<1,1>>>(); break;
      case 3: debugMark<3><<<1,1>>>(); break;
      case 4: debugMark<4><<<1,1>>>(); break;
      case 5: debugMark<5><<<1,1>>>(); break;
      case 6: debugMark<6><<<1,1>>>(); break;
      case 7: debugMark<7><<<1,1>>>(); break;
      case 8: debugMark<8><<<1,1>>>(); break;
      case 9: debugMark<9><<<1,1>>>(); break;
      default: 
         break;
   };

}
#else
void debugMark(int t) {if (0==t) debugMark<0><<<1,1>>>();};
#endif
